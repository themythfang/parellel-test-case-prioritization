#include "hip/hip_runtime.h"
#include "non_dom_sort.h"
#include "initdata.h"
#include "initpop.h"
#include "gene.h"
#include "father_select.h"
#include "crossover.h"
#include "mutation.h"
#include "fitness.h"
#include "elism_select.h"
#include "time_record.h"
#include "cuda_initdata.h"
#include "cuda_fitness.h"
#include "cuda_crossover.h"
#include <iostream>
using namespace std;


// @description: Antoniol's Single Point Crossover
// generate (pop_num / 2) 

#define CHAR_BIT_NUM 8

// Two assuption:
// 1. $TEST_NUM is the power of 2;
// 2. $TEST_NUM is less or equal to 512.
__device__ void CGetSonGene(int* c_gene_map, int* c_pos_map, int p, int test_num, int pop_num) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int bdim = MAXTHREAD;

  for (int i = 0; i < test_num; i += bdim) {
    if (tid + i < test_num) {
      int gene_code = c_pos_map[p * test_num + tid + i];
      c_gene_map[(pop_num + bid) * test_num + tid + i] = c_gene_map[p * test_num + gene_code];
    }
  }
  __syncthreads();
}

__global__ void CCudaCross(int* c_gene_map, int* c_pos_map, int* c_father_vec, int test_num, int pop_num) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int bdim = MAXTHREAD;

  int p = c_father_vec[bid];
  __syncthreads();

  CGetSonGene(c_gene_map, c_pos_map, p, test_num, pop_num);

  __syncthreads();
}

void* ScanChooseCroPoint_C() {
  vector<int> flags;
  vector<int> pos;
  flags.resize(test_num);
  pos.resize(test_num);

  int t;

  // Set the random seed.
  int scan_num = test_num * cross_propo;

  for (int i = 0; i < pop_num; ++i) {
    // Initialize the vector with value 0
    flags.assign(test_num, 0);
    for (int j = 0; j < scan_num; ++j) {
      t = rand() % test_num;
      while (flags[t] == 1) {
	t = rand() % test_num;
      }
      flags[t] = 1;
    }

    // Generation the new generation using the algorithm "SCAN".
    int count = 0;
    for (int j = 0; j < test_num; ++j) {
      pos[j] = count;
      if (flags[j] == 1) {
	++count;
      }
    }

    for (int j = 0; j < test_num; ++j) {
      pop_vec[pop_num + i].flags_vec[j] = pos[j];
    }

    for (int j = 0; j < test_num; ++j) {
      if (flags[j] == 0) {
	pos[j] = scan_num + j - pos[j];
      }
    }

    for (int j = 0; j < test_num; ++j) {
      t_pos_map[test_num * i + j] = pos[j];
      //      pop_vec[pop_num + i].pos_vec[j] = pos[j];
    }
  }
}

void CudaScanCross(TimeRecord& c, CudaTimeRecord& time_rec, CudaTimeRecord& time_in_rec, CudaTimeRecord& time_out_rec) {
  c.Start();
  ScanChooseCroPoint_C();
  c.Record();

  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = pop_vec[i].gene_code_vec[j];
    }
  }

  for (int i = 0; i < pop_num; ++i) {
    t_father_vec[i] = father_sel_vec[i];
  }


  time_in_rec.Start();
  hipGetErrorString(hipMemcpy(c_gene_map, t_gene_map, pop_num * test_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_pos_map, t_pos_map, pop_num * test_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_father_vec, t_father_vec, pop_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  time_in_rec.Record();
  //  time_out_rec.Record();

  cout << "ScanCross kernel start" << endl;
  time_rec.Start();
  CCudaCross<<<pop_num, MAXTHREAD>>>(c_gene_map, c_pos_map, c_father_vec, test_num, pop_num);
  time_rec.Record();

  cout << "ScanCross kernel end" << endl;

  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = 0;
    }
  }

  // Copy result out.
  time_out_rec.Start();
  hipGetErrorString(hipMemcpy(t_gene_map, c_gene_map,
    				pop_num * test_num * 2 * sizeof(int), hipMemcpyDeviceToHost));
  time_out_rec.Record();


  /*
  for (int i = 0; i < pop_num; ++i) {
    int father = t_father_vec[i];
    for (int j = 0; j < test_num; ++j) {
      cout << pop_vec[father].gene_code_vec[j] << " ";
    }
    cout << endl;
    for (int j = 0; j < test_num; ++j) {
      cout << t_gene_map[(pop_num + i) * test_num + j] << " ";
    }
    cout << endl;
    cout << endl;
    getchar();
  }

  for (int i = 0; i < pop_num * 2; ++i) {
    int count = 0;

    for (int j = 0; j < test_num; ++j) {
      cout << t_gene_map[i * test_num + j] << " ";
      count += t_gene_map[i * test_num + j];
    }

    cout << "count: " << count << endl;
    cout << endl;
    getchar();
  }
*/
  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < test_num; ++j) {
      pop_vec[i].gene_code_vec[j] = t_gene_map[i * test_num + j];
    }
  }
}
