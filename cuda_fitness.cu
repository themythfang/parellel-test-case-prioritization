#include "hip/hip_runtime.h"
#include "cuda_fitness.h"
#include "cuda_initdata.h"
#include "initdata.h"
#include "time_record.h"
#include "cuda_time_record.h"

#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>

__device__ void CCompFit(int bid, int tid, int bdim, int* s_total_sta, int* s_min_exe,
			 float* c_time_vec, int* c_gene_map, float* c_fitness_vec, int test_num, int code_num) {
  for (int i = (bdim / 2); i > 0; i /= 2) {
    if (tid < i && tid + i < code_num) {
      s_total_sta[tid] += s_total_sta[tid + i];
      s_min_exe[tid] = max(s_min_exe[tid], s_min_exe[tid + i]);
    }
    __syncthreads();
  }
  __syncthreads();

  __shared__ float s_exe_time_vec[MAXTHREAD];
  s_exe_time_vec[tid] = 0;
  __syncthreads();

  int min_num = s_min_exe[0];
  __syncthreads();

  for (int i = 0; i <= min_num; i += bdim) {
    if (tid + i <= min_num) {
      int gene_code = c_gene_map[bid * test_num + tid + i];
      s_exe_time_vec[tid] += c_time_vec[gene_code];
    }
  }

  //  for (int i = (bdim / 2); i > 0 && i <= min_num ; i /= 2) {
  for (int i = (bdim / 2); i > 0 ; i /= 2) {
    if (tid < i) {
      s_exe_time_vec[tid] += s_exe_time_vec[tid + i];
    }
  }

  if (tid == 0) {
    c_fitness_vec[bid * 2] = (1.0f - s_total_sta[0] * 1.0f / (code_num * test_num) + 1.0f / (test_num << 1));
    //    c_fitness_vec[bid * 2] = s_total_sta[0];
    c_fitness_vec[bid * 2 + 1] = s_exe_time_vec[0];
    //    c_fitness_vec[bid * 2 + 1] = min_num;
    //    c_fitness_vec[bid * 2 + 1] = c_gene_map[bid * test_num + 50];
    //    c_fitness_vec[bid * 2 + 1] = min_num;
    //    c_fitness_vec[bid * 2 + 1] = 0;
  }

  __syncthreads();
}

__global__ void CInitFit(char* c_cover_map, float* c_time_vec, int* c_gene_map, 
                         int* c_first_exe_map, float* c_fitness_vec, int test_num, int code_num) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int bdim = MAXTHREAD;

  __shared__ int s_min_exe[MAXTHREAD];
  s_min_exe[tid] = MIN_NUM;
  __syncthreads();

  __shared__ int s_total_sta[MAXTHREAD];
  s_total_sta[tid] = 0;
  __syncthreads();

  int gene_code;
  int first;
  for(int i = 0; i < code_num; i += bdim) {
    if((tid + i) < code_num) {
      first = test_num;
      for (int j = 0; j < test_num; ++j) {
	gene_code = c_gene_map[bid * test_num + j];
	if (c_cover_map[gene_code * code_num + tid + i] == '1') {
	  first = j;
	  break;
	}
      }
      if (first == test_num) {
	// Uncoverd.
	c_first_exe_map[bid * code_num + tid + i] = NOEXIST;
      } else {
	c_first_exe_map[bid * code_num + tid + i] = first;
	s_total_sta[tid] += (first + 1);
	s_min_exe[tid] = max(s_min_exe[tid], first);
      }
      __syncthreads();
    }
  }
  __syncthreads();

  CCompFit(bid, tid, bdim, s_total_sta, s_min_exe,
    	   c_time_vec, c_gene_map, c_fitness_vec, test_num, code_num);

  __syncthreads();
}

void CudaCompFit(CudaTimeRecord& time_rec, int com_num) {
  for (int i = 0; i < com_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = pop_vec[i].gene_code_vec[j];
    }
  }

  for (int i = 0; i < com_num; ++i) {
    for (int j = 0; j < code_num; ++j) {
      t_first_exe_map[i * code_num + j] = pop_vec[i].first_exe_vec[j];
    }
  }

  time_rec.Start();

  hipGetErrorString(hipMemcpy(c_gene_map, t_gene_map, pop_num * test_num * 2 * sizeof(int), 
                                hipMemcpyHostToDevice));

  cout << "initfit kernel start" << endl;
  CInitFit<<<com_num, MAXTHREAD>>>(c_cover_map, c_time_vec, c_gene_map, c_first_exe_map, c_fitness_vec, test_num, code_num);

  cout << "muta kernel end" << endl;

  // Copy result out.
  hipGetErrorString(hipMemcpy(t_first_exe_map, c_first_exe_map,
				pop_num * code_num * 2 * sizeof(int), hipMemcpyDeviceToHost));
  hipGetErrorString(hipMemcpy(t_fitness_vec, c_fitness_vec, pop_num * 4 * sizeof(float),
                                hipMemcpyDeviceToHost));

  time_rec.Record();

  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < code_num; ++j) {
      pop_vec[i].first_exe_vec[j] = t_first_exe_map[i * code_num + j];
    }

    pop_vec[i].apsc = t_fitness_vec[i * 2];
    pop_vec[i].eff_time = t_fitness_vec[i * 2 + 1];
  }
}

void CudaCompFit2(CudaTimeRecord& time_rec, CudaTimeRecord& time_in_rec, CudaTimeRecord& time_out_rec, int com_num) {
  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = pop_vec[pop_num + i].gene_code_vec[j];
    }
  }


  time_in_rec.Start();
  hipGetErrorString(hipMemcpy(c_gene_map, t_gene_map, pop_num * test_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  time_in_rec.Record();

  cout << "initfit kernel start" << endl;
  time_rec.Start();
  CInitFit<<<pop_num, MAXTHREAD>>>(c_cover_map, c_time_vec, c_gene_map, c_first_exe_map, c_fitness_vec, test_num, code_num);
  time_rec.Record();

  cout << "muta kernel end" << endl;
  time_out_rec.Start();
  hipGetErrorString(hipMemcpy(t_fitness_vec, c_fitness_vec, pop_num * 4 * sizeof(float),
                                hipMemcpyDeviceToHost));
  time_out_rec.Record();


  for (int i = 0; i < pop_num; ++i) {
    pop_vec[pop_num + i].apsc = t_fitness_vec[i * 2];
    pop_vec[pop_num + i].eff_time = t_fitness_vec[i * 2 + 1];
  }
}
