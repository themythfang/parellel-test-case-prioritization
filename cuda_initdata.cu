#include <iostream>
#include "cuda_initdata.h"
#include "initdata.h"
#include "gene.h"
#include "time_record.h"

#include <cstdlib>
#include <hip/hip_runtime.h>

float* c_time_vec = NULL;
int* c_gene_map = NULL;
int* c_first_exe_map = NULL;
int *c_rand_vec = NULL;
int *c_father_vec = NULL;
float *c_fitness_vec = NULL;
int* c_single_pos_map = NULL;
int* c_swap_vec = NULL;
char* c_cover_map = NULL;
int* c_is_cover_vec = NULL;
int* c_pos_map = NULL;

// Temp var.
float * t_time_vec = NULL;
int* t_gene_map = NULL;
int* t_first_exe_map = NULL;
int *t_rand_vec = NULL;
int *t_father_vec = NULL;
float* t_fitness_vec = NULL;
int* t_single_pos_map = NULL;
int* t_swap_vec = NULL;
char* t_cover_map = NULL;
int* t_is_cover_vec = NULL;
int* t_pos_map = NULL;

// @description: initialize the data nsga2 need
// @return
// 0 success
// -1 failed to read data
// int CudaInitData(TimeRecord& c);
int CudaInitData(TimeRecord& c) {
  cout << "int CudaInitData(TimeRecord& c) {" << endl;
  getchar();
  if (InitData() != 0) {
    return -1;
  }
  srand(time(NULL));
  // Temp var.
  t_time_vec = new float[test_num];
  t_gene_map = new int[pop_num * test_num * 2];
  t_first_exe_map = new int[pop_num * code_num * 2];
  t_rand_vec = new int[pop_num];
  t_father_vec = new int[pop_num];
  t_fitness_vec = new float[pop_num * 4];
  t_single_pos_map = new int[pop_num * test_num];
  t_swap_vec = new int[pop_num * 4];
  t_cover_map = new char[test_num * code_num];
  t_is_cover_vec = new int[code_num];
  t_pos_map = new int[pop_num * test_num];

  for (int i = 0; i < test_num; ++i) {
    t_time_vec[i] = gen_vec[i].time;
  }

  for (int i = 0; i < test_num; ++i) {
    for (int j = 0; j < code_num; ++j) {
      t_cover_map[i * code_num + j] = gen_vec[i].coverage[j];
    }
  }

  for (int i = 0; i < code_num; ++i) {
    t_is_cover_vec[i] = 0;
  }
  for (int i = 0; i < code_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      if (gen_vec[j].coverage[i] == '1') {
        t_is_cover_vec[i] = 1;
        break;
      }
    }
  }

  c.Start();
  CUDA_CALL(hipMalloc((void**)&c_time_vec, test_num * sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&c_gene_map, pop_num * test_num * 2 * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_first_exe_map, pop_num * code_num * 2 * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_rand_vec, pop_num * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_father_vec, pop_num * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_fitness_vec, (pop_num * 4) * sizeof(float)));
  CUDA_CALL(hipMalloc((void**)&c_single_pos_map, pop_num * test_num * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_swap_vec, pop_num * 4 * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_cover_map, test_num * code_num * sizeof(char)));
  CUDA_CALL(hipMalloc((void**)&c_is_cover_vec, code_num * sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&c_pos_map, pop_num * test_num * sizeof(int)));

  hipGetErrorString(hipMemcpy(c_time_vec, t_time_vec, test_num * sizeof(float),
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_cover_map, t_cover_map, test_num * code_num * sizeof(char),
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_is_cover_vec, t_is_cover_vec, code_num * sizeof(int),
                                hipMemcpyHostToDevice));
  c.Record();

  return 0;
}

void CudaDeleteData() {
  t_time_vec = new float[test_num];
  t_gene_map = new int[pop_num * test_num * 2];
  t_first_exe_map = new int[pop_num * code_num * 2];
  t_rand_vec = new int[pop_num];
  t_father_vec = new int[pop_num];
  t_fitness_vec = new float[pop_num * 4];
  t_single_pos_map = new int[pop_num * test_num];
  t_swap_vec = new int[pop_num * 4];
  t_cover_map = new char[test_num * code_num];
  t_is_cover_vec = new int[code_num];
  t_pos_map = new int[pop_num * test_num];

  delete []t_time_vec;
  delete []t_gene_map;
  delete []t_first_exe_map;
  delete []t_rand_vec;
  delete []t_father_vec;
  delete []t_fitness_vec;
  delete []t_single_pos_map;
  delete []t_swap_vec;
  delete []t_cover_map;
  delete []t_is_cover_vec;
  delete []t_pos_map;

  hipFree(c_time_vec);
  hipFree(c_gene_map);
  hipFree(c_first_exe_map);
  hipFree(c_rand_vec);
  hipFree(c_father_vec);
  hipFree(c_fitness_vec);
  hipFree(c_swap_vec);
  hipFree(c_cover_map);
  hipFree(c_pos_map);
  hipFree(c_single_pos_map);
  hipFree(c_is_cover_vec);
}
