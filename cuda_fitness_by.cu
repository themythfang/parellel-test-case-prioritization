#include "hip/hip_runtime.h"
#include "cuda_fitness.h"
#include "cuda_initdata.h"
#include "initdata.h"
#include "time_record.h"
#include "cuda_time_record.h"

#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <>


__global__ void CInitialFit(float* c_time_vec, int* c_gene_map, 
			    int* c_first_exe_map, float* c_fitness_vec) {

}

__global__ void CSingleCrossUpdFit(float* c_time_vec, int* c_gene_map, 
			     int* c_first_exe_map, int* c_rand_vec, 
			     int* c_father_vec, float* c_fitness_vec, 
			     int* c_single_pos_map) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int bdim = MAXTHREAD;

  int f1, f2;
  if (bid % 2 == 0) {
    f1 = c_father_vec[bid];
    f2 = c_father_vec[bid + 1];
  } else {
    f1 = c_father_vec[bid];
    f2 = c_father_vec[bid - 1];
  }
  __syncthreads();

  int k = c_rand_vec[bid / 2];
  __syncthreads();

  __shared__ int s_min_exe[MAXTHREAD];
  s_min_exe[tid] = MIN_NUM;
  __syncthreads();

  __shared__ int s_total_sta[MAXTHREAD];
  s_total_sta[tid] = 0;
  __syncthreads();

  for(int i = 0; i < CODE_NUM; i += bdim) {
    if((tid + i) < CODE_NUM) {
      int t = c_first_exe_map[f1 * CODE_NUM + tid + i];
      if (t >= k) {
	int first_code = c_first_exe_map[f2 * CODE_NUM + tid + i];
        t = c_single_pos_map[bid * TEST_NUM + first_code];
      }
      c_first_exe_map[(POP_NUM + bid) * CODE_NUM + tid + i] = t;
      if (t != NOEXIST) {
        s_total_sta[tid] += (t + 1);
        s_min_exe[tid] = max(s_min_exe[tid], t);
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

void CudaSinglePointCrossoverUpdFit(CudaTimeRecord& time_rec) {
  // void CudaSinglePointCrossoverUpdFit(TimeRecord& time_rec) {
  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = pop_vec[pop_num + i].gene_code_vec[j];
    }
  }
  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < code_num; ++j) {
      t_first_exe_map[i * code_num + j] = pop_vec[i].first_exe_vec[j];
    }
  }

  for (int i = 0; i < pop_num; ++i) {
    t_father_vec[i] = father_sel_vec[i];
  }
  for (int i = 0; i < (pop_num >> 1); ++i) {
    t_rand_vec[i] = single_rand_vec[i];
  }
  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_single_pos_map[i * test_num + j] = pop_vec[pop_num + i].single_pos_vec[j];
    }
  }

  time_rec.Start();

  hipGetErrorString(hipMemcpy(c_gene_map, t_gene_map, pop_num * test_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_first_exe_map, t_first_exe_map, pop_num * code_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_father_vec, t_father_vec, pop_num * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_rand_vec, t_rand_vec, (pop_num >> 1) * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_single_pos_map, t_single_pos_map, pop_num * test_num * sizeof(int), 
				hipMemcpyHostToDevice));
  CSingleCrossUpdFit<<<pop_num, MAXTHREAD>>>
    (c_time_vec, c_gene_map, c_first_exe_map, 
     c_rand_vec, c_father_vec, c_fitness_vec, 
     c_single_pos_map);

  // Copy result out.
  hipGetErrorString(hipMemcpy(t_first_exe_map, c_first_exe_map + pop_num * code_num, 
				pop_num * code_num * sizeof(int), hipMemcpyDeviceToHost));
  hipGetErrorString(hipMemcpy(t_fitness_vec, c_fitness_vec, (pop_num << 1) * sizeof(float),
                                hipMemcpyDeviceToHost));

  time_rec.Record();

  for (int i = 0; i < pop_num; ++i) {
    for (int j = 0; j < code_num; ++j) {
      pop_vec[pop_num + i].first_exe_vec[j] = t_first_exe_map[i * code_num + j];
    }

    pop_vec[pop_num + i].apsc = t_fitness_vec[i * 2];
    pop_vec[pop_num + i].eff_time = t_fitness_vec[i * 2 + 1];
  }
}

//__device__ void CCompFit(int bid, int tid, int bdim, int* s_total_sta, int* s_min_exe,
//			 float* c_time_vec, int* c_gene_map, float* c_fitness_vec) {
__device__ void CCompFit(int bid, int tid, int bdim, volatile int* s_total_sta, volatile int* s_min_exe,
			 volatile float* c_time_vec, volatile int* c_gene_map, volatile float* c_fitness_vec) {

  for (int i = (bdim / 2); i > 0; i /= 2) {
    if (tid < i && tid + i < CODE_NUM) {
      s_total_sta[tid] += s_total_sta[tid + i];
      s_min_exe[tid] = max(s_min_exe[tid], s_min_exe[tid + i]);
    }
    __syncthreads();
  }
  __syncthreads();

  __shared__ float s_exe_time_vec[MAXTHREAD];
  s_exe_time_vec[tid] = 0;
  __syncthreads();

  int min_num = s_min_exe[0];
  __syncthreads();

  //  int gene_code;
  for (int i = 0; i < min_num; i += bdim) {
    /*
    gene_code = c_gene_map[bid * TEST_NUM + tid + i];
    s_exe_time_vec[tid] += c_time_vec[gene_code];
    */
    s_exe_time_vec[tid] += c_time_vec[c_gene_map[bid * TEST_NUM + tid + i]];
  }

  for (int i = (bdim / 2); i > 0; i /= 2) {
    if (tid < i && tid + i < min_num) {
      s_exe_time_vec[tid] += s_exe_time_vec[tid + i];
    }
  }
  /*
  if (tid == 0) {
    //    c_fitness_vec[bid * 2] = (1.0f - s_total_sta[0] * 1.0f / (CODE_NUM * TEST_NUM) + 1.0f / (TEST_NUM << 1));
    //    c_fitness_vec[bid * 2] = s_total_sta[0];
    c_fitness_vec[bid * 2] = -3;
    //    c_fitness_vec[bid * 2 + 1] = s_exe_time_vec[0];
    //    c_fitness_vec[bid * 2 + 1] = c_gene_map[bid * TEST_NUM + 50];
    //    c_fitness_vec[bid * 2 + 1] = min_num;
    c_fitness_vec[bid * 2 + 1] = bid;
  }
  */
  __syncthreads();
}

__global__ void CSwapMutationUpdFit(char* c_cover_map, float* c_time_vec, 
				    int* c_gene_map, int* c_first_exe_map, 
				    float* c_fitness_vec, int* c_swap_vec) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int bdim = MAXTHREAD;

  __shared__ int s_min_exe[MAXTHREAD];
  s_min_exe[tid] = MIN_NUM;
  __syncthreads();

  __shared__ int s_total_sta[MAXTHREAD];
  s_total_sta[tid] = 0;
  __syncthreads();

  int a = c_swap_vec[bid * 2];
  int b = c_swap_vec[bid * 2 + 1];
  __syncthreads();

  for(int i = 0; i < CODE_NUM; i += bdim) {
    if((tid + i) < CODE_NUM) {
      int first = c_first_exe_map[bid * CODE_NUM + tid + i];
      if (first != NOEXIST) {
	int gene_code;
	if (b > first) {
	  if (a == first) {
	    for (int j = a; j <= b; ++j) {
	      gene_code = c_gene_map[bid * TEST_NUM + j];
	      if (c_cover_map[gene_code * CODE_NUM + tid + i] == '1') {
		first = j;
		break;
	      }
	    }
	  } else if (a < first) {
	    gene_code = c_gene_map[bid * TEST_NUM + a];
	    if (c_cover_map[gene_code * CODE_NUM + tid + i] == '1') {
	      first = a;
	    }
	  }
	} else if (b == first) {
	  first = a;
	} else {
	  // NOTHING todo.
	}
      }
      c_first_exe_map[bid * CODE_NUM + tid + i] = first;
      s_total_sta[tid] += (first + 1);
      s_min_exe[tid] = max(s_min_exe[tid], first);
    }
    __syncthreads();
  }
  __syncthreads();

  if (tid == 0) {
    c_fitness_vec[bid * 2] = a;
    c_fitness_vec[bid * 2 + 1] = b;
  }
  __syncthreads();

  CCompFit(bid, tid, bdim, s_total_sta, s_min_exe,
  	   c_time_vec, c_gene_map, c_fitness_vec);

  __syncthreads();
}

void CudaSwapMutationUpdFit(CudaTimeRecord& time_rec) {
  //void CudaSwapMutationUpdFit(TimeRecord& time_rec) {
  cout << "start 0" << endl;
  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < test_num; ++j) {
      t_gene_map[i * test_num + j] = pop_vec[i].gene_code_vec[j];
    }
  }

  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < code_num; ++j) {
      t_first_exe_map[i * code_num + j] = pop_vec[i].first_exe_vec[j];
    }
  }

  for (int i = 0; i < pop_num * 2; ++i) {
    t_swap_vec[i * 2] = swap_vec_a[i];
    t_swap_vec[i * 2 + 1] = swap_vec_b[i];
  }

  cout << "time_rec" << endl;
  time_rec.Start();

  hipGetErrorString(hipMemcpy(c_gene_map, t_gene_map, pop_num * test_num * 2 * sizeof(int), 
                                hipMemcpyHostToDevice));
  // TODO: May be duzzy
  hipGetErrorString(hipMemcpy(c_first_exe_map, t_first_exe_map, pop_num * code_num * 2 * sizeof(int), 
                                hipMemcpyHostToDevice));
  hipGetErrorString(hipMemcpy(c_swap_vec, t_swap_vec, pop_num * 4 * sizeof(int), 
				hipMemcpyHostToDevice));

  cout << "muta kernel start" << endl;
  CSwapMutationUpdFit<<<pop_num * 2, MAXTHREAD>>>(c_cover_map, c_time_vec, c_gene_map, c_first_exe_map, c_fitness_vec, c_swap_vec);

  cout << "muta kernel end" << endl;

  // Copy result out.
  hipGetErrorString(hipMemcpy(t_first_exe_map, c_first_exe_map,
				pop_num * code_num * 2 * sizeof(int), hipMemcpyDeviceToHost));
  hipGetErrorString(hipMemcpy(t_fitness_vec, c_fitness_vec, pop_num * 4 * sizeof(float),
                                hipMemcpyDeviceToHost));

  time_rec.Record();

  for (int i = 0; i < pop_num * 2; ++i) {
    for (int j = 0; j < code_num; ++j) {
      pop_vec[i].first_exe_vec[j] = t_first_exe_map[i * code_num + j];
    }

    pop_vec[i].apsc = t_fitness_vec[i * 2];
    pop_vec[i].eff_time = t_fitness_vec[i * 2 + 1];
  }
}
